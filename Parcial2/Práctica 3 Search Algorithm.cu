#include "hip/hip_runtime.h"
//Práctica 3: Search Algorithm   ----    Jonathan Cuevas 0225174
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <random>

using namespace std;

void bubbleSort_host(int* a, int size) {
    for (int i = 0; i < size - 1; i++) {
        for (int j = 0; j < size - i - 1; j++) {
            if (a[j] > a[j + 1]) {
                int aux = a[j + 1];
                a[j + 1] = a[j];
                a[j] = aux;
            }
        }
    }
}
__global__ void search(int* a, int size, int* pos, int target) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        if (a[tid] == target) {
            *pos = tid;
        }
    }
}
int main() {
    int size = 128, target = 77;
    int* dev_a, * dev_pos;
    int* host_a = (int*)malloc(size * sizeof(int));
    int* res = (int*)malloc(size * sizeof(int));
    int* pos = (int*)malloc(sizeof(int));
    pos[0] = -1;
    
    hipMalloc(&dev_a, size * sizeof(int));
    hipMalloc(&dev_pos, sizeof(int));

    printf("Arreglo:\n");
    for (int i = 0; i < size; i++) {
        int r1 = rand() % (128);
        host_a[i] = r1;
        printf("%d ", host_a[i]);
    }
    printf("\n");

    hipMemcpy(dev_a, host_a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_pos, pos, sizeof(int), hipMemcpyHostToDevice);

    dim3 grid(1);
    dim3 block(1024);
    search << <grid, block >> > (dev_a, size, dev_pos, target);
    hipDeviceSynchronize();

    hipMemcpy(pos, dev_pos, sizeof(int), hipMemcpyDeviceToHost);

    if(pos[0] == -1){
        printf("No está en la lista\n");
    }
    else{
        printf("El numero %d está en el indice %d \n",target, pos[0]);
    }

    free(host_a);
    free(pos);
    free(res);
    hipFree(dev_a);
    hipFree(dev_pos);

    return 0;
}