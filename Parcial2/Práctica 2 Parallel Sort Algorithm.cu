#include "hip/hip_runtime.h"
//Práctica 2: Parallel Sort Algorithm   ----    Jonathan Cuevas 0225174
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <random>

using namespace std;

void bubbleSort_host(int* a, int size) {
    for (int i = 0; i < size-1; i++) {
        for (int j = 0; j < size-i - 1; j++) {
            if (a[j] > a[j + 1]) {
                int aux = a[j + 1];
                a[j + 1] = a[j];
                a[j] = aux;
            }
        }
    }
}

__global__ void bubbleSort_dev(int* a, int size) {
    int tid = threadIdx.x;
    for (int i = 0; i < size; i++) {
        int offset = i % 2;
        int leftSide = 2 * tid + offset;
        int rightSide = leftSide + 1;
        if (rightSide < size) {
            if (a[leftSide] > a[rightSide]) {
                int aux = a[leftSide];
                a[leftSide] = a[rightSide];
                a[rightSide] = aux;
            }
        }
        __syncthreads();
    }
}

int main() {
    int Array = 1024;
    int* host_config, * res;
    int* dev_config;
    host_config = (int*)malloc(Array * sizeof(int));
    res = (int*)malloc(Array * sizeof(int));
    hipMalloc(&dev_config, Array * sizeof(Array));

    printf("Arreglo original: \n");
    for (int i = 0; i < Array; i++) {
        int r1 = (rand() % (1024));
        host_config[i] = r1;
        printf("%d ", host_config[i]);
    }
    printf("\n");

    dim3 grid(1);
    dim3 block(Array);

    clock_t gpu_start, gpu_stop;
    gpu_start = clock();
    hipMemcpy(dev_config, host_config, Array * sizeof(int), hipMemcpyHostToDevice);
    bubbleSort_dev << <grid, block >> > (dev_config, Array);
    gpu_stop = clock();
    double cps_gpu = (double)((double)(gpu_stop - gpu_start) / CLOCKS_PER_SEC);

    printf("\n\nBubble sort CPU: \n");
    for (int i = 0; i < Array; i++) {
        printf("%d ", host_config[i]);
    }
    printf("\nTIEMPO DE EJECUCION CPU: %4.6f \n\r", cps_gpu);

    clock_t gpu_start1, gpu_stop1;
    gpu_start1 = clock();
    hipMemcpy(res, dev_config, Array * sizeof(int), hipMemcpyDeviceToHost);
    bubbleSort_host(host_config, Array);
    gpu_stop1 = clock();
    cps_gpu = (double)((double)(gpu_stop1 - gpu_start1) / CLOCKS_PER_SEC);
    
    printf("\n\nBubble sort GPU\n");
    for (int i = 0; i < Array; i++) {
        printf("%d ", res[i]);
    }
    printf("\nTIEMPO DE EJECUCION GPU: %4.6f \n\r", cps_gpu);
    return 0;
}