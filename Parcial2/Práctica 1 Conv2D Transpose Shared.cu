#include "hip/hip_runtime.h"
//Práctica 1: Conv2D Transpose Shared Memory 	---	Jonathan Cuevas 0225174
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <random>

using namespace std;

__global__ void matrizTranspose(int* a, int* b, int n) {
    __shared__ int s[64];
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int offsetBlock = blockIdx.x * blockDim.x * blockDim.y;
    int offsetRow = blockIdx.y * blockDim.x * blockDim.y * gridDim.x;
    int gid = tid + offsetBlock + offsetRow;

    int row = gid / n;
    int col = gid - row * n;
    if (gid < n * n) {
        s[row * n + col] = a[row * n + col];
        __syncthreads();
        b[col * n + row] = s[row * n + col];
    }
}

__global__ void convolucion(int* a, int* b, int* k, int n, int m, int kernelSize) {
    __shared__ int s[64];
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int offsetBlock = blockIdx.x * blockDim.x * blockDim.y;
    int offsetRow = blockIdx.y * blockDim.x * blockDim.y * gridDim.x;
    int gid = tid + offsetBlock + offsetRow;

    int rowActual = gid / n;
    int colActual = gid - rowActual * n;

    int suma = 0;
    for (int i = -1; i <= 1; i++) {
        for (int j = -1; j <= 1; j++) {
            if (rowActual + i >= 0 && rowActual + i < n && colActual + j >= 0 && colActual + j < n) {
                s[(i + 1) * kernelSize + j + 1] = k[(i + 1) * kernelSize + j + 1];
                __syncthreads();
                suma += a[(rowActual + i) * m + colActual + j] * s[(i + 1) * kernelSize + j + 1];
            }
        }
    }
    b[rowActual * m + colActual] = suma;
}

int main() {
    const int kernelSize = 5, row = 8, col = 8;
    int* host_aKernel, * host_convKernel, * host_a, * host_b;
    int* dev_aKernel, * dev_convKernel, * dev_a, * dev_b;
    host_aKernel = (int*)malloc(kernelSize * kernelSize * sizeof(int));
    host_convKernel = (int*)malloc(kernelSize * kernelSize * sizeof(int));
    host_a = (int*)malloc(row * col * sizeof(int));
    host_b = (int*)malloc(row * col * sizeof(int));

    hipMalloc(&dev_aKernel, kernelSize * kernelSize * sizeof(int));
    hipMalloc(&dev_convKernel, kernelSize * kernelSize * sizeof(int));
    hipMalloc(&dev_a, row * col * sizeof(int));
    hipMalloc(&dev_b, row * col * sizeof(int));

    for (int i = 0; i < kernelSize * kernelSize; i++) {
        int r1 = rand() % (1);
        host_aKernel[i] = r1;
        host_convKernel[i] = 0;
    }

    for (int i = 0; i < (row* col); i++) {
        int r1 = rand() % (10);
        host_a[i] = r1;
        host_b[i] = 0;
    }

    host_aKernel[3] = 1;
    printf("Kernel 5x5: \n");
    for (int i = 0; i < kernelSize; i++) {
        for (int j = 0; j < kernelSize; j++) {
            printf("%d ", host_aKernel[i * kernelSize + j]);
        }
        printf("\n");
    }

    printf("\nMatriz A: \n");
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            printf("%d ", host_a[i * col + j]);
        }
        printf("\n");
    }

    hipMemcpy(dev_aKernel, host_aKernel, kernelSize * kernelSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_convKernel, host_convKernel, kernelSize * kernelSize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_a, host_a, row*col*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, row*col*sizeof(int), hipMemcpyHostToDevice);

    dim3 block(32, 32);
    dim3 grid(32 / (kernelSize * kernelSize), 32 / (kernelSize * kernelSize));
    matrizTranspose << <grid, block >> > (dev_aKernel, dev_convKernel, kernelSize);
    hipMemcpy(host_convKernel, dev_convKernel, kernelSize * kernelSize * sizeof(int), hipMemcpyDeviceToHost);

    cout << "\nRes Kernel:\n";
    for (int i = 0; i < kernelSize; i++) {
        for (int j = 0; j < kernelSize; j++) {
            cout << host_convKernel[i * kernelSize + j] << " ";
        }
        cout << "\n";
    }

    dim3 block2(32, 32);
    dim3 grid2((64 + (row*col) - 1) / (row *col), (64 + (row *col) - 1) / (row*col));
    convolucion << <grid2, block2 >> > (dev_a, dev_b, dev_convKernel, row, col, kernelSize);
    hipMemcpy(host_b, dev_b, row*col*sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipDeviceReset();


    printf("\nMatriz B (Convolución): \n");
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            printf("%d ", host_b[i * col + j]);
        }
        printf("\n");
    }

    free(host_aKernel);
    free(host_convKernel);

    hipFree(dev_aKernel);
    hipFree(dev_convKernel);

    return 0;
}