#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <time.h>

using namespace std;

__global__ void multiplication(int* a, int* b, int* result){
    int id = threadIdx.x;
    result[id] = a[id]*b[id];
}

__global__ void printKernel() {
    printf("threadIdx %d %d %d \n", threadIdx.x, threadIdx.y, threadIdx.z);
    printf("blockIdx %d %d %d \n", blockIdx.x, blockIdx.y, blockIdx.z);
    printf("gridDim %d %d %d \n", gridDim.x, gridDim.y, gridDim.z);
}

int main(){
    const int n = 3;
    int size = n * sizeof(n);

    int vectorA[n] = { 2, 7, 10 };
    int vectorB[n] = { 4, 0, 1 };
    int result[n] = { 0, 0, 0 };

    int* devA = 0;
    int* devB = 0;
    int* devC = 0;

    hipMalloc((void**)&devA, size);
    hipMalloc((void**)&devB, size);
    hipMalloc((void**)&devC, size);

    hipMemcpy(devA, vectorA, size, hipMemcpyHostToDevice);
    hipMemcpy(devB, vectorB, size, hipMemcpyHostToDevice);
    hipMemcpy(devC, result, size, hipMemcpyHostToDevice);

    multiplication << <1, n >> > (devA, devB, devC);
    hipDeviceSynchronize();

    hipMemcpy(result, devC, size, hipMemcpyDeviceToHost);
    printf("Vector A: {%d, %d, %d}\n", vectorA[0], vectorA[1], vectorA[2]);
    printf("Vector B: {%d, %d, %d}\n", vectorB[0], vectorB[1], vectorB[2]);
    printf("Resultado multiplicación: {%d, %d, %d}\n", result[0], result[1], result[2]);
    hipDeviceReset();

    hipFree(devA);
    hipFree(devB);
    hipFree(devC);

    dim3 block(2, 2, 2);
    dim3 grid(4 / block.x, 4 / block.y, 4 / block.z);
    printKernel << <grid, block >> > ();

    return 0;
}