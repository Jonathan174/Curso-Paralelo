#include "hip/hip_runtime.h"
//Práctica 5: Matrix Transpose Unrolling Complete   ----    Jonathan Cuevas 0225174
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <random>

using namespace std;

__global__ void unrollingTranspose(int* a, int* b, int size) {
    int gid = (threadIdx.x + threadIdx.y * blockDim.x) + (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y);
    int offset = blockDim.x / 2;

    for (int i = 0; i < (size* size + blockDim.x * blockDim.y - 1) / (blockDim.x * blockDim.y); i += 2)
    {
        if (gid + blockDim.x * blockDim.y * i < size * size) {
            b[(gid % size * size + gid / size) + offset * i] = a[gid + blockDim.x * blockDim.y * i];
        }
        if (gid + blockDim.x * blockDim.y * i + blockDim.x * blockDim.y < size * size) {
            b[(gid % size * size + gid / size) + offset * i + offset] = a[gid + blockDim.x * blockDim.y * i + blockDim.x * blockDim.y];
        }
    }
}

int main() {
    const int size = 16;
    int* host_a, * host_b;
    int* dev_a, * dev_b;
    host_a = (int*)malloc(size * size * sizeof(int));
    host_b = (int*)malloc(size * size * sizeof(int));
    hipMalloc(&dev_a, size * size * sizeof(int));
    hipMalloc(&dev_b, size * size * sizeof(int));

    for (int i = 0; i < size * size; i++) {
        int r1 = rand() % (256);
        host_a[i] = r1;
        host_b[i] = 0;
    }

    printf("\nMatriz original: \n");
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            printf("%d ", host_a[i * size + j]);
        }
        printf("\n");
    }

    hipMemcpy(dev_a, host_a, size * size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, size * size * sizeof(int), hipMemcpyHostToDevice);

    dim3 grid(1);
    dim3 block(4, 4);
    unrollingTranspose << <1, block >> > (dev_a, dev_b, size);
    hipMemcpy(host_b, dev_b, size * size * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipDeviceReset();

    printf("\nMatriz transpuesta: \n");
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            printf("%d ", host_b[i * size + j]);
        }
        printf("\n");
    }

    free(host_a);
    free(host_b);
    hipFree(dev_a);
    hipFree(dev_b);

    return 0;
}